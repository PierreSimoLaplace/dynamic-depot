#include<iostream>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<iomanip>
#include<hipblas.h>
#include"gemmgpu.h"
#include"gemmcpu.h"
#define ROW 16
using namespace std;
void matMulBlasGpu(const Matrix TempA,const Matrix TempB,Matrix TempC,float &elapsedTime);
int main(){
	struct timeval beg,end;
	srand(time(NULL));
	Matrix A;//row-major order
	A.width=4;A.height=2;A.stride=A.width;
	A.elements=(float *)malloc(A.width*A.height*sizeof(float));
	Matrix B;//row-major order
	B.width=2;B.height=A.width;B.stride=B.width;
	B.elements=(float *)malloc(B.width*B.height*sizeof(float));
	Matrix C;//row-major order
	C.width=B.width;C.height=A.height;C.stride=C.width;
	C.elements=(float *)malloc(C.width*C.height*sizeof(float));

	cout<<"A:"<<endl;
	for(int i=0;i<A.height;i++){
		cout<<i+1<<"th row:"<<endl;
		for(int j=0;j<A.width;j++){
			//A.elements[i*A.width+j]=10*(1.0*rand()/RAND_MAX)-5;
			A.elements[i*A.width+j]=rand()%10-5;	
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<A.elements[i*A.width+j];
			if((j+1)%ROW==0 && j<A.width-1)
				cout<<endl;
			else if(j==A.width-1)
				cout<<endl;
		}
	}
	cout<<endl;

	cout<<"B:"<<endl;
	for(int i=0;i<B.height;i++){
		cout<<i+1<<"th row:"<<endl;
		for(int j=0;j<B.width;j++){
			//B.elements[i*B.width+j]=10*(1.0*rand()/RAND_MAX)-5;
			B.elements[i*B.width+j]=rand()%10-5;	
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<B.elements[i*B.width+j];
			if((j+1)%ROW==0 && j<B.width-1)
				cout<<endl;
			else if(j==B.width-1)
				cout<<endl;
		}
	}
	cout<<endl;

	
	float elapsedTime=0;
	matMulGpu(A,B,C,elapsedTime);
	cout<<"Time consumption on GPU:"<<elapsedTime<<" ms"<<endl;
	cout<<"C:"<<endl;
	for(int i=0;i<C.height;i++){
		cout<<i+1<<"th row:"<<endl;
		for(int j=0;j<C.width;j++){
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<C.elements[i*C.width+j];
			if((j+1)%ROW==0 && j<C.width-1)
				cout<<endl;
			else if(j==C.width-1)
				cout<<endl;
		}
	}
	cout<<endl;

	for(int i=0;i<C.width*C.height;i++)
		C.elements[i]=0;
	gettimeofday(&beg,NULL);
	matMulCpu(A,B,C);
	gettimeofday(&end,NULL);
	cout<<"Time consumption on CPU:"<<float((end.tv_sec-beg.tv_sec)*1000+1.0*(end.tv_usec-beg.tv_usec)/1000)<<" ms"<<endl;

	cout<<"C:"<<endl;
	for(int i=0;i<C.height;i++){
		cout<<i+1<<"th row:"<<endl;
		for(int j=0;j<C.width;j++){
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<C.elements[i*C.width+j];
			if((j+1)%ROW==0 && j<C.width-1)
				cout<<endl;
			else if(j==C.width-1)
				cout<<endl;
		}
	}
	cout<<endl;
	
	
	Matrix TempA;//column-major order
	TempA.height=A.height;TempA.width=A.width;TempA.stride=TempA.width;
	TempA.elements=(float *)malloc(TempA.width*TempA.height*sizeof(float));
	Matrix TempB;//column-major order
	TempB.height=B.height;TempB.width=B.width;TempB.stride=TempB.width;
	TempB.elements=(float *)malloc(TempB.width*TempB.height*sizeof(float));
	Matrix TempC;//column-major order
	TempC.height=C.height;TempC.width=C.width;TempC.stride=TempC.width;
	TempC.elements=(float *)malloc(TempC.width*TempC.height*sizeof(float));

	cout<<"TempA:"<<endl;
	for(int j=0;j<TempA.width;j++){
		cout<<j+1<<"th column:"<<endl;
		for(int i=0;i<TempA.height;i++){
			TempA.elements[j*TempA.height+i]=A.elements[i*TempA.width+j];
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<TempA.elements[j*TempA.height+i];
			if((i+1)%ROW==0 && i<TempA.height-1)
				cout<<endl;
			else if(i==TempA.height-1)
				cout<<endl;
		}
		cout<<endl;
	}
	cout<<endl;

	cout<<"TempB:"<<endl;
	for(int j=0;j<TempB.width;j++){
		cout<<j+1<<"th column:"<<endl;
		for(int i=0;i<TempB.height;i++){
			TempB.elements[j*TempB.height+i]=B.elements[i*TempB.width+j];
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<TempB.elements[j*TempB.height+i];
			if((i+1)%ROW==0 && i<TempB.height-1)
				cout<<endl;
			else if(i==TempB.height-1)
				cout<<endl;
		}
	}
	cout<<endl;

	for(int i=0;i<C.width*C.height;i++)
		C.elements[i]=0;

	elapsedTime=0;
	matMulBlasGpu(TempA,TempB,TempC,elapsedTime);
	cout<<"Time consumption using CUBLAS:"<<elapsedTime<<" ms"<<endl;
	cout<<"C:"<<endl;
	for(int i=0;i<C.height;i++){
		cout<<i+1<<"th row:"<<endl;
		for(int j=0;j<C.width;j++){
			C.elements[i*C.width+j]=TempC.elements[j*C.height+i];
			cout<<fixed<<setprecision(2)<<setiosflags(ios::right)<<setw(6)<<C.elements[i*C.width+j];
			if((j+1)%ROW==0 && j<C.width-1)
				cout<<endl;
			else if(j==C.width-1)
				cout<<endl;
		}
	}
	cout<<endl;
	
	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}
void matMulBlasGpu(const Matrix TempA,const Matrix TempB,Matrix TempC,float &elapsedTime){
	float alpha=1,beta=0;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status=hipblasCreate(&handle);

	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);
	hipEventRecord(beg,0);

	float *d_TempA,*d_TempB,*d_TempC;
	hipMalloc((void **)&d_TempA,TempA.height*TempA.width*sizeof(float));
	hipMalloc((void **)&d_TempB,TempB.height*TempB.width*sizeof(float));
	hipMalloc((void **)&d_TempC,TempC.height*TempC.width*sizeof(float));

	hipblasSetVector(TempA.width*TempA.height,sizeof(float),TempA.elements,1,d_TempA,1);
	hipblasSetVector(TempB.width*TempB.height,sizeof(float),TempB.elements,1,d_TempB,1);

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,TempA.height,TempB.width,TempA.width,&alpha,d_TempA,TempA.height,d_TempB,TempB.height,&beta,d_TempC,TempA.height);
	hipDeviceSynchronize();

	hipblasGetVector(TempC.height*TempC.width,sizeof(float),d_TempC,1,TempC.elements,1);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipFree(d_TempA);
	hipFree(d_TempB);
	hipFree(d_TempC);
	hipEventDestroy(beg);
	hipEventDestroy(end);
	hipblasDestroy(handle);
}
