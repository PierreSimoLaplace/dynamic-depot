#include "hip/hip_runtime.h"
#include"gemmgpu.h"

/* API:Matrix Multiplication worked by GPU */
void matMulGpu(const Matrix A,const Matrix B,Matrix C,float &elapsedTime){
	size_t size;
	Matrix d_A;
	d_A.width=A.width;d_A.height=A.height;d_A.stride=A.stride;
	Matrix d_B;
	d_B.width=B.width;d_B.height=B.height;d_B.stride=B.stride;
	Matrix d_C;
	d_C.width=C.width;d_C.height=C.height;d_C.stride=C.stride;

	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);
	hipEventRecord(beg,0);

	size=d_A.width*d_A.height*sizeof(float);
	hipMalloc(&d_A.elements,size);
	hipMemcpy(d_A.elements,A.elements,size,hipMemcpyHostToDevice);

	size=d_B.width*d_B.height*sizeof(float);
	hipMalloc(&d_B.elements,size);
	hipMemcpy(d_B.elements,B.elements,size,hipMemcpyHostToDevice);

	size=d_C.width*d_C.height*sizeof(float);
	hipMalloc(&d_C.elements,size);

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((d_B.width+dimBlock.x-1)/dimBlock.x,(d_A.height+dimBlock.y-1)/dimBlock.y);
	matMulGpuKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C);
	
	hipMemcpy(C.elements,d_C.elements,size,hipMemcpyDeviceToHost);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipEventDestroy(beg);
	hipEventDestroy(end);

	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

/* Builtin:Matrix Multiplication kernel workde by GPU */
__global__ void matMulGpuKernel(const Matrix d_A,const Matrix d_B,Matrix d_C){
	int blockWidth=blockIdx.x;
	int blockHeight=blockIdx.y;
	float sum=0;
	int width=threadIdx.x;
	int height=threadIdx.y;

	__shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

	Matrix Csub=getSubMatrix(d_C,blockHeight,blockWidth);

	for(int num=0;num<(d_A.width+BLOCK_SIZE-1)/BLOCK_SIZE-1;num++){
		Matrix Asub=getSubMatrix(d_A,blockHeight,num);
		Matrix Bsub=getSubMatrix(d_B,num,blockWidth);

		if((num*BLOCK_SIZE+width)<d_A.width && (blockHeight*BLOCK_SIZE+height)<d_A.height)
			sharedA[height][width]=getElement(Asub,height,width);
		if((blockWidth*BLOCK_SIZE+width)<d_B.width && (num*BLOCK_SIZE+height)<d_B.height)
			sharedB[height][width]=getElement(Bsub,height,width);
		
		__syncthreads();
		
		for(int index=0;index<BLOCK_SIZE;index++)
			sum+=sharedA[height][index]*sharedB[index][width];

		__syncthreads();
	}
	{
		int num=(d_A.width+BLOCK_SIZE-1)/BLOCK_SIZE-1;
		Matrix Asub=getSubMatrix(d_A,blockHeight,num);
		Matrix Bsub=getSubMatrix(d_B,num,blockWidth);

		if((num*BLOCK_SIZE+width)<d_A.width && (blockHeight*BLOCK_SIZE+height)<d_A.height)
			sharedA[height][width]=getElement(Asub,height,width);
		if((blockWidth*BLOCK_SIZE+width)<d_B.width && (num*BLOCK_SIZE+height)<d_B.height)
			sharedB[height][width]=getElement(Bsub,height,width);
		
		__syncthreads();

		if((blockHeight*BLOCK_SIZE+height)<d_A.height && (blockWidth*BLOCK_SIZE+width)<d_B.width){
			for(int index=0,length=d_A.width%BLOCK_SIZE;index<(length>0?length:BLOCK_SIZE);index++)
				sum+=sharedA[height][index]*sharedB[index][width];
		}

		__syncthreads();

		if((blockHeight*BLOCK_SIZE+height)<d_A.height && (blockWidth*BLOCK_SIZE+width)<d_B.width)
			setElement(Csub,height,width,sum);
	}

}

/* Builtin:To get certain matrix element */
__device__ float getElement(const Matrix M,int height,int width){
	return M.elements[height*M.stride+width];
}

/* Builtin:To set certain matrix element */
__device__ void setElement(Matrix M,int height,int width,float value){
	M.elements[height*M.stride+width]=value;
}

/* Builtin:To get submatrix of original matrix */
__device__ Matrix getSubMatrix(const Matrix M,int height,int width){
	Matrix Msub;
	Msub.width=BLOCK_SIZE;Msub.height=BLOCK_SIZE;Msub.stride=M.stride;
	Msub.elements=&M.elements[height*Msub.height*Msub.stride+width*Msub.width];
	return Msub;
}
