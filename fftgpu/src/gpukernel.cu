#include "hip/hip_runtime.h"
#include"gpukernel.h"

/* API:Fourier Transform 1d,h_n1 must be 2^int */
void fft1d(fcomplexcuda *h_input,int h_n1,float &elapsedTime){
	int h_p1=0;
	float h_PI=4*atan(1);
	int h_2n1=h_n1/2;
	while(h_n1>pow(2,h_p1)){
		h_p1++;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_n1),&h_n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n1),&h_2n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p1),&h_p1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(PI),&h_PI,sizeof(float));

	fcomplexcuda *d_input;
	hipMalloc((void **)&d_input,h_n1*sizeof(fcomplexcuda));

	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((h_n1+dimBlock.x-1)/dimBlock.x);
	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);

	hipEventRecord(beg,0);
	hipMemcpy(d_input,h_input,h_n1*sizeof(fcomplexcuda),hipMemcpyHostToDevice);
	fft1dKernel<<<dimGrid,dimBlock>>>(d_input);
	hipMemcpy(h_input,d_input,h_n1*sizeof(fcomplexcuda),hipMemcpyDeviceToHost);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipEventDestroy(beg);
	hipEventDestroy(end);

	hipFree(d_input);
}

/* Builtin:Kernel function of Fourier Transform 1d */
__global__ void fft1dKernel(fcomplexcuda *d_input){
	__shared__ int p2k,p2kAdd1;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int sign;
	int indexLeft;
	fcomplexcuda w;
	if(idx<d_n1){
		for(int k=0;k<p1;k++){
			if(threadIdx.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idx%p2kAdd1<p2k?1:-1;
			w=omega(idx%p2k,p2kAdd1);
			indexLeft=idx/p2kAdd1*p2k+idx%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx]=d_input[indexLeft]+w*d_input[indexLeft+d_2n1];
			//__syncthreads();
			__threadfence();
		}
	}

}

/* API:Inverse Fourier Transform 1d,h_n1 must be 2^int */
void ifft1d(fcomplexcuda *h_input,int h_n1,float &elapsedTime){
	int h_p1=0;
	float h_PI=4*atan(1);
	int h_2n1=h_n1/2;
	while(h_n1>pow(2,h_p1)){
		h_p1++;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_n1),&h_n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n1),&h_2n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p1),&h_p1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(PI),&h_PI,sizeof(float));

	fcomplexcuda *d_input;
	hipMalloc((void **)&d_input,h_n1*sizeof(fcomplexcuda));

	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((h_n1+dimBlock.x-1)/dimBlock.x);
	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);

	hipEventRecord(beg,0);
	hipMemcpy(d_input,h_input,h_n1*sizeof(fcomplexcuda),hipMemcpyHostToDevice);
	ifft1dKernel<<<dimGrid,dimBlock>>>(d_input);
	hipMemcpy(h_input,d_input,h_n1*sizeof(fcomplexcuda),hipMemcpyDeviceToHost);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipEventDestroy(beg);
	hipEventDestroy(end);

	hipFree(d_input);
}

/* Builtin:Kernel function of Inverse Fourier Transform 1d */
__global__ void ifft1dKernel(fcomplexcuda *d_input){
	__shared__ int p2k,p2kAdd1;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int sign;
	int indexLeft;
	fcomplexcuda w;
	if(idx<d_n1){
		for(int k=0;k<p1;k++){
			if(threadIdx.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idx%p2kAdd1<p2k?1:-1;
			w=iomega(idx%p2k,p2kAdd1);
			indexLeft=idx/p2kAdd1*p2k+idx%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx]=d_input[indexLeft]+w*d_input[indexLeft+d_2n1];
			//__syncthreads();
			__threadfence();
		}
		d_input[idx].real/=d_n1;
		d_input[idx].imag/=d_n1;
	}

}

/* API:Fourier Transform 2d,h_n1 and h_n2 must be 2^int */
void fft2d(fcomplexcuda *h_input,int h_n1,int h_n2,float &elapsedTime){
	int h_p1=0,h_p2=0;
	float h_PI=4*atan(1);
	int h_2n1=h_n1/2;
	int h_2n2=h_n2/2;
	while(h_n1>pow(2,h_p1)){
		h_p1++;
	}
	while(h_n2>pow(2,h_p2)){
		h_p2++;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_n1),&h_n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_n2),&h_n2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n1),&h_2n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n2),&h_2n2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p1),&h_p1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p2),&h_p2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(PI),&h_PI,sizeof(float));

	fcomplexcuda *d_input;
	hipMalloc((void **)&d_input,h_n1*h_n2*sizeof(fcomplexcuda));

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((h_n1+dimBlock.x-1)/dimBlock.x,(h_n2+dimBlock.y-1)/dimBlock.y);
	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);

	hipEventRecord(beg,0);
	hipMemcpy(d_input,h_input,h_n1*h_n2*sizeof(fcomplexcuda),hipMemcpyHostToDevice);
	fft2dKernel<<<dimGrid,dimBlock>>>(d_input);
	hipMemcpy(h_input,d_input,h_n1*h_n2*sizeof(fcomplexcuda),hipMemcpyDeviceToHost);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipEventDestroy(beg);
	hipEventDestroy(end);

	hipFree(d_input);
}

/* Builtin:Kernel function of Fourier Transform 2d */
__global__ void fft2dKernel(fcomplexcuda *d_input){
	__shared__ int p2k,p2kAdd1;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int idy=threadIdx.y+blockIdx.y*blockDim.y;
	int stridey=d_n1*idy;
	int sign;
	int indexLeft;
	fcomplexcuda w;
	if(idx<d_n1 && idy<d_n2){
		for(int k=0;k<p1;k++){
			if(threadIdx.x+threadIdx.y*blockDim.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idx%p2kAdd1<p2k?1:-1;
			w=omega(idx%p2k,p2kAdd1);
			indexLeft=idx/p2kAdd1*p2k+idx%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx+stridey]=d_input[indexLeft+stridey]+w*d_input[indexLeft+d_2n1+stridey];
			//__syncthreads();
			__threadfence();
		}
		//__syncthreads();
		__threadfence();
		for(int k=0;k<p2;k++){
			if(threadIdx.x+threadIdx.y*blockDim.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idy%p2kAdd1<p2k?1:-1;
			w=omega(idy%p2k,p2kAdd1);
			indexLeft=idy/p2kAdd1*p2k+idy%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx+stridey]=d_input[indexLeft*d_n1+idx]+w*d_input[(indexLeft+d_2n2)*d_n1+idx];
			//__syncthreads();
			__threadfence();
		}
	}

}

/* API:Inverse Fourier Transform 2d,h_n1 and h_n2 must be 2^int */
void ifft2d(fcomplexcuda *h_input,int h_n1,int h_n2,float &elapsedTime){
	int h_p1=0,h_p2=0;
	float h_PI=4*atan(1);
	int h_2n1=h_n1/2;
	int h_2n2=h_n2/2;
	while(h_n1>pow(2,h_p1)){
		h_p1++;
	}
	while(h_n2>pow(2,h_p2)){
		h_p2++;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_n1),&h_n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_n2),&h_n2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n1),&h_2n1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_2n2),&h_2n2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p1),&h_p1,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p2),&h_p2,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(PI),&h_PI,sizeof(float));

	fcomplexcuda *d_input;
	hipMalloc((void **)&d_input,h_n1*h_n2*sizeof(fcomplexcuda));

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((h_n1+dimBlock.x-1)/dimBlock.x,(h_n2+dimBlock.y-1)/dimBlock.y);
	hipEvent_t beg,end;
	hipEventCreate(&beg);
	hipEventCreate(&end);

	hipEventRecord(beg,0);
	hipMemcpy(d_input,h_input,h_n1*h_n2*sizeof(fcomplexcuda),hipMemcpyHostToDevice);
	ifft2dKernel<<<dimGrid,dimBlock>>>(d_input);
	hipMemcpy(h_input,d_input,h_n1*h_n2*sizeof(fcomplexcuda),hipMemcpyDeviceToHost);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,beg,end);

	hipEventDestroy(beg);
	hipEventDestroy(end);

	hipFree(d_input);
}

/* Builtin:Kernel function of Inverse Fourier Transform 2d */
__global__ void ifft2dKernel(fcomplexcuda *d_input){
	__shared__ int p2k,p2kAdd1;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int idy=threadIdx.y+blockIdx.y*blockDim.y;
	int stridey=d_n1*idy;
	int sign;
	int indexLeft;
	fcomplexcuda w;
	if(idx<d_n1 && idy<d_n2){
		for(int k=0;k<p1;k++){
			if(threadIdx.x+threadIdx.y*blockDim.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idx%p2kAdd1<p2k?1:-1;
			w=iomega(idx%p2k,p2kAdd1);
			indexLeft=idx/p2kAdd1*p2k+idx%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx+stridey]=d_input[indexLeft+stridey]+w*d_input[indexLeft+d_2n1+stridey];
			//__syncthreads();
			__threadfence();
		}
		d_input[idx+stridey].real/=d_n1;
		d_input[idx+stridey].imag/=d_n1;
		//__syncthreads();
		__threadfence();

		for(int k=0;k<p2;k++){
			if(threadIdx.x+threadIdx.y*blockDim.x==0){
				p2k=dPow(2,k);
				p2kAdd1=dPow(2,k+1);
			}
			__syncthreads();
			sign=idy%p2kAdd1<p2k?1:-1;
			w=iomega(idy%p2k,p2kAdd1);
			indexLeft=idy/p2kAdd1*p2k+idy%p2k;
			w=fcomplexcuda(sign*w.real,sign*w.imag);
			d_input[idx+stridey]=d_input[indexLeft*d_n1+idx]+w*d_input[(indexLeft+d_2n2)*d_n1+idx];
			//__syncthreads();
			__threadfence();
		}
		d_input[idx+stridey].real/=d_n2;
		d_input[idx+stridey].imag/=d_n2;
	}

}

/* Builtin:Calculate ith unit root in fft */
__device__ fcomplexcuda omega(int i,int N){
	fcomplexcuda w;
	w.real=cos(2*PI*i/N);
	w.imag=sin(-2*PI*i/N);
	return w;
}

/* Builtin:Calculate ith unit root in ifft */
__device__ fcomplexcuda iomega(int i,int N){
	fcomplexcuda w;
	w.real=cos(2*PI*i/N);
	w.imag=sin(2*PI*i/N);
	return w;
}

/* Builtin:Calculate x^p fast */
__device__ int dPow(int x,int p){
	int res=1;
	while(p){
		if(p&1)
			res*=x;
		p>>=1;
		x*=x;
	}
	return res;
}
